#include "hip/hip_runtime.h"
/***********************************************************
  > File Name: add_bias_relu.cu
  > Author: stx
  > Mail: 18817608993@163.com
  > Created Time: 2023年11月12日 星期日 23时53分04秒
  > Modified Time:2023年11月12日 星期日 23时53分04秒
 *******************************************************/

// grid b
// block M
template <typename Type>
__global__ void AddBiasReluKernel0(const Type *x, const Type *bias, Type *y, const int N)
{
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  for (auto id = tid; id < N; id += blockDim.x) {
    float v = (float)x[bid * N + id] + (float)bias[id];
    v = fmax(v, 0);
    y[bid * N + id] = (Type)v;
  }
}

int AddBiasRelu0(const float *x, const float *bias, float *y, const int B, const int N, hipStream_t stream)
{
  dim3 grid(B);
  dim3 block(std::min(N, 1024));
  AddBiasReluKernel0<float><<<gird, block, 0, stream>>>(x, bias, y, N);
  if (hipGetLastError() != hipSuccess) {
    LogErr("launch kernel failed");
    return -1;
  }
  return 0;
}

template <typename Type>
__global__ void AddBiasReluKernel1(const Type *x, const Type *bias, Type *y, const int Num, const int N)
{
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < Num) {
    int bias_id = id % N;
    float v = (float)x[id] + (float)bias[bias_id];
    v = fmax(v, 0);
    y[id] = (Type)v;
  }
}

int AddBiasRelu1(const float *x, const float *bias, float *y, const int B, const int N, hipStream_t stream) {
  int Num = B * N;
  dim3 block(std::min(Num, 1024));
  dim3 grid((num + block - 1) / block);
  AddBiasReluKernel1<float><<<grid, block, 0, stream>>>(x, bias, y, Num, N);
  if (hipGetLastError() != hipSuccess) {
    LogErr("launch kernel failed");
    return -1;
  }
  return 0;
}

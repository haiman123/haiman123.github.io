/***********************************************************
  > File Name: hello.cu
  > Author: stx
  > Mail: 18817608993@163.com
  > Created Time: 2023年11月10日 星期五 01时16分38秒
  > Modified Time:2023年11月10日 星期五 01时16分38秒
 *******************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <vector>
__global__ void HelloWorldKernel() { printf("[BlockIdx[%d, %d, %d], threadIdx:[%d, %d, %d] hello world\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z); }

int Test1() {
   // cudaSetDevice(1);
   dim3 grid(2, 1, 1);
   dim3 block{8, 1, 1};
   HelloWorldKernel<<<grid, block>>>();
   hipDeviceSynchronize();
   return 0;
}

__global__ void VecAddKernel(const float *A, const float *B, float *C, const int N) {
//__global__ void VecAddKernel(const float *A, const float *B, float *C) {
  auto bid = blockIdx.x;
  auto tid = threadIdx.x;
  auto id = bid * blockDim.x + tid;
  // printf("A[%d}:%f\n", tid, A[tid]);
  if (id < N) 
    C[id] = A[id] + B[id];
  
}

int Test2() {
  // C = A + B
  int N = 200000;
  float *A(nullptr);
  float *B(nullptr);
  float *C(nullptr);
  hipMalloc(&A, sizeof(float) * N);
  hipMalloc(&B, sizeof(float) * N);
  hipMalloc(&C, sizeof(float) * N);
  std::vector<float> cpu_a(N, 0);
  std::vector<float> cpu_b(N, 0);
  std::vector<float> cpu_c(N, 0);
  for (std::size_t i = 0; i != cpu_a.size(); i++) {
    cpu_a[i] = 0.1f * i;
    cpu_b[i] = 0.2f * i;
    cpu_c[i] = cpu_a[i] + cpu_b[i];
  }
  hipMemcpyAsync(A, cpu_a.data(), sizeof(float) * N, hipMemcpyHostToDevice, nullptr);
  hipMemcpyAsync(B, cpu_b.data(), sizeof(float) * N, hipMemcpyHostToDevice, nullptr);
  
  // printf("A ptr:%lld", (intptr_t)A);
  dim3 grid((N + 1023) / 1024, 1, 1);
  dim3 block(1024, 1, 1); // max 1024 thread
  hipStream_t stream;
  hipStreamCreate(& stream);
  VecAddKernel<<<grid, block, 0, stream>>>(A, B, C, N);
  // printf("============cuda error is %d\n", cudaGetLastError());
  if (hipGetLastError() != hipSuccess) {
    printf("launch kernel failed! \n");
    return -1;
  }
  hipDeviceSynchronize();
  std::vector<float> copy_cpu_c(N, 0);
  hipMemcpy(copy_cpu_c.data(), C, sizeof(float) * N, hipMemcpyDeviceToHost);
  for (std::size_t i = 0; i != copy_cpu_c.size(); i++) {
    if (copy_cpu_c[i] != cpu_c[i]) {
      printf("result %d error\n", i);
      return -1;
    }
    // printf("%f + %f = c[%d] =  %f\n", cpu_a[i], cpu_b[i], i, copy_cpu_c[i]);

  }
  printf("Test Passed\n");
  return 0;
}

int main() {
  Test1();
  Test2();
  return 0;
};

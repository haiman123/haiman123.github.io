#include "hip/hip_runtime.h"
/***********************************************************
  > File Name: transpose.cu
  > Author: stx
  > Mail: 18817608993@163.com
  > Created Time: 2023年11月19日 星期日 22时45分04秒
  > Modified Time:2023年11月19日 星期日 22时45分04秒
 *******************************************************/

__global__ void TransposeKernel1(const int *x, const int *y, const int M, const ing N)
{
  __shared__ float data[32][32];
  int row_back = blockIdx.y * blockDim.y;
  int col_base = blockIdx.x * blockDim.x;
  int ty = threadIdx.y;
  int tx = threadIdx.x;
  int row_id = row_base + ty;
  int col_id = col_back + tx;
  data[ty][tx] = (row_id < M && col_id < N) ? x[row_id * N + col_id] : 0.0f;
  row_id = row_base + tx;
  col_id = col_base + ty;
  if (row_id < M && col_id < N) {
    y[col_id * M + row_id] = data[tx][ty];
  }
}

__global__ void TransposeKernel0(const int *x, const int *y, const int N, const int M)
{
  int row_id = blockIdx.y * blockDim.y + threadIdx.y;
  int col_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (row_id < M && col_id < N) {
    y[col_id * M + row_id] = x[row_id * N + col_id];
  }
}

int TransPoseFun0(const float *x, const float *y, const int M, const int N, hipStream_t stream)
{
  dim3 block(32, 32);
  dim3 grid((N+31) >> 5, (M + 31) >> 5, 1);
  TransposeKernel0<<<grid, block, 0, stream>>>(x, y, M, N);
  if (hipGetLastError() != hipSuccess) {
    LogErr("call kernel failed");
    return -1;
  }
  return 0;
}
